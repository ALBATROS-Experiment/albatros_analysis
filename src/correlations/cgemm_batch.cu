#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include <stdio.h>
//nvcc -Xcompiler -fPIC -shared -o libcgemm_batch.so cgemm_batch.cu -lcublas

extern "C"
void cgemm_strided_batched(
    const hipComplex* A,    // ptr to batch of A, shape (M,K,batch), column‑major
    const hipComplex* B,    // ptr to batch of B, shape (N,K,batch), column‑major
    hipComplex*       C,    // ptr to batch of C, shape (M,N,batch), column‑major
    int M, int N, int K,
    int batchCount)
{
    // leading dims in column‑major
    int lda = M, ldb = N, ldc = M;
    // strides between consecutive batches
    long long strideA = (long long)M * K;
    long long strideB = (long long)N * K;
    long long strideC = (long long)M * N;

    // Scalars
    const hipComplex α = make_hipComplex(1.0f, 0.0f);
    const hipComplex β = make_hipComplex(0.0f, 0.0f);

    // cuBLAS handle
    // int vnum;
    hipblasHandle_t h;
    if (hipblasCreate(&h) != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "hipblasCreate failed\n");
        return;
    }
    // cublasGetVersion(h, &vnum);
    // printf("CuBLAS version number %d\n", vnum);
    // Perform: C = α·A·Bᴴ + β·C  (batched)
    hipblasStatus_t stat = hipblasCgemmStridedBatched(
        h,
        HIPBLAS_OP_N,      // A not transposed
        HIPBLAS_OP_C,      // B conjugate‑transposed
        M,                // #rows of A and C
        N,                // #cols of Bᴴ and C
        K,                // #cols of A == #rows of B
        &α,
        A, lda, strideA,
        B, ldb, strideB,
        &β,
        C, ldc, strideC,
        batchCount);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "hipblasCgemmStridedBatched failed: %d\n", stat);
    }
    hipblasDestroy(h);
}