//nvcc -o libpycufft.so pycufft.cu -shared -lcufft -Xcompiler -fPIC -lgomp

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
/*--------------------------------------------------------------------------------*/
const char* _cufftGetErrorEnum( hipfftResult_t error )
{
    switch ( error )
    {
        case HIPFFT_SUCCESS:
        return "HIPFFT_SUCCESS";

        case HIPFFT_INVALID_PLAN:
        return "cuFFT was passed an invalid plan handle\n";

        case HIPFFT_ALLOC_FAILED:
        return "cuFFT failed to allocate GPU or CPU memory\n";

        // No longer used
        case HIPFFT_INVALID_TYPE:
        return "HIPFFT_INVALID_TYPE\n";

        case HIPFFT_INVALID_VALUE:
        return "User specified an invalid pointer or parameter\n";

        case HIPFFT_INTERNAL_ERROR:
        return "Driver or internal cuFFT library error\n";

        case HIPFFT_EXEC_FAILED:
        return "Failed to execute an FFT on the GPU\n";

        case HIPFFT_SETUP_FAILED:
        return "The cuFFT library failed to initialize\n";

        case HIPFFT_INVALID_SIZE:
        return "User specified an invalid transform size\n";

        // No longer used
        case HIPFFT_UNALIGNED_DATA:
        return "HIPFFT_UNALIGNED_DATA\n";

        case HIPFFT_INCOMPLETE_PARAMETER_LIST:
        return "Missing parameters in call\n";

        case HIPFFT_INVALID_DEVICE:
        return "Execution of a plan was on different GPU than plan creation\n";

        case HIPFFT_PARSE_ERROR:
        return "Internal plan database error\n";

        case HIPFFT_NO_WORKSPACE:
        return "No workspace has been provided prior to plan execution\n";

        case HIPFFT_NOT_IMPLEMENTED:
        return "HIPFFT_NOT_IMPLEMENTED\n";

        case CUFFT_LICENSE_ERROR:
        return "CUFFT_LICENSE_ERROR\n";
    }

    return "<unknown>";
}

extern "C"{
void cufft_r2c_mohan(hipfftComplex *out, float * data, int nrows, int ncols, hipfftHandle * plan_ptr)
{
    // nrows = rows in input data; ncols = columns in input data.
    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipDeviceSynchronize();
    // hipEventRecord(start, 0);
    hipError_t hipError_t;

        hipError_t = hipGetLastError();

        if( hipError_t != hipSuccess )

        {

        fprintf(stderr, "CUDA Runtime API Error reported : %s\n", hipGetErrorString(hipError_t));

        exit(EXIT_FAILURE);

        }
    if (plan_ptr)
    {
        //if a plan is available execute and move on
        // printf("Using user's plan r2c\n");
        // hipEventRecord(start, 0);
        hipfftResult_t error;
        error=hipfftExecR2C(*plan_ptr,data,out);
        if (error!=HIPFFT_SUCCESS)
        {
            fprintf(stderr,"R2C (%d, %d) FAILED\n", nrows, ncols);
            fprintf(stderr,_cufftGetErrorEnum(error));
            exit(EXIT_FAILURE);
        }
        // hipDeviceSynchronize();
    }
    else
    {
        hipfftHandle plan;
        if (hipfftPlan1d(&plan,ncols,HIPFFT_R2C,nrows)!=HIPFFT_SUCCESS)
            fprintf(stderr,"Error planning dft\n");
        if (hipfftExecR2C(plan,data,out)!=HIPFFT_SUCCESS)
            fprintf(stderr,"Error executing dft manual r2c\n");
        // hipDeviceSynchronize();
        if (hipfftDestroy(plan)!= HIPFFT_SUCCESS)
            fprintf(stderr,"Error destroying plan.\n");
    }
    // hipDeviceSynchronize();
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop);
    // printf("r2c took %12.4g\n",milliseconds);
}
void cufft_c2c(hipfftComplex *out, hipfftComplex * data, int nrows, int ncols, int direction, hipfftHandle * plan_ptr)
{
    hipError_t hipError_t;
    hipError_t = hipGetLastError();
    if( hipError_t != hipSuccess )
    {
        fprintf(stderr, "CUDA Runtime API Error reported : %s\n", hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }
    direction = (direction == -1) ? HIPFFT_FORWARD : HIPFFT_BACKWARD;
    if (plan_ptr)
    {
        //if a plan is available execute and move on
        // printf("Using user's plan r2c\n");
        // hipEventRecord(start, 0);
        hipfftResult_t error;
        error=hipfftExecC2C(*plan_ptr,data,out,direction);
        if (error!=HIPFFT_SUCCESS)
        {
            fprintf(stderr,"C2C (%d, %d) FAILED\n", nrows, ncols);
            fprintf(stderr,_cufftGetErrorEnum(error));
            exit(EXIT_FAILURE);
        }
        // hipDeviceSynchronize();
    }
    else
    {
        hipfftHandle plan;
        if (hipfftPlan1d(&plan,ncols,HIPFFT_C2C,nrows)!=HIPFFT_SUCCESS)
            fprintf(stderr,"Error planning dft\n");
        if (hipfftExecC2C(plan,data,out,direction)!=HIPFFT_SUCCESS)
            fprintf(stderr,"Error executing dft manual r2c\n");
        // hipDeviceSynchronize();
        if (hipfftDestroy(plan)!= HIPFFT_SUCCESS)
            fprintf(stderr,"Error destroying plan.\n");
    }
}
void cufft_c2r_mohan(float *out, hipfftComplex * data, int nrows, int ncols, hipfftHandle * plan_ptr)
{
    // nrows = rows in output data; ncols = columns in output data.
    // if (plan==NULL)
    // {
    //     printf("plan is null\n");
    //     printf("nrows = %d ncols = %d\n", nrows, ncols);
    // }
    hipError_t hipError_t;

        hipError_t = hipGetLastError();

        if( hipError_t != hipSuccess )

        {

        fprintf(stderr, "CUDA Runtime API Error reported : %s\n", hipGetErrorString(hipError_t));

        exit(EXIT_FAILURE);

        }
    if (plan_ptr)
    {
        // printf("Using user's plan c2r\n");
        //if a plan is available execute and move on
        if (hipfftExecC2R(*plan_ptr,data,out)!=HIPFFT_SUCCESS)
            fprintf(stderr,"Error executing dft c2r\n");
        // hipDeviceSynchronize();
    }
    else
    {  
        hipfftHandle plan;
        if (hipfftPlan1d(&plan,ncols,HIPFFT_C2R,nrows)!=HIPFFT_SUCCESS)
            fprintf(stderr,"Error planning dft\n");
        if (hipfftExecC2R(plan,data,out)!=HIPFFT_SUCCESS)
            fprintf(stderr,"Error executing dft manual c2r\n");
        // hipDeviceSynchronize();
        if (hipfftDestroy(plan)!= HIPFFT_SUCCESS)
            fprintf(stderr,"Error destroying plan.\n");
    }
}
/*--------------------------------------------------------------------------------*/
void get_plan_r2c(int nrows, int ncols, hipfftHandle *plan, size_t * work_size)
{
    // nrows, ncols of input
    // Initialize an empty plan.
    hipError_t hipError_t;

    hipError_t = hipGetLastError();

    if( hipError_t != hipSuccess )

    {

    fprintf(stderr, "CUDA Runtime API Error reported : %s\n", hipGetErrorString(hipError_t));

    exit(EXIT_FAILURE);

    }
    hipfftCreate(plan);
    // Turn off auto-allocation by default. This must be done before the
    // plan is actually created, so we can't use the shortcut hipfftPlan1d
    hipfftResult_t error;
    error=hipfftSetAutoAllocation(*plan, 0);
    if(error!=HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT setAutoAllocation returned %d", error);
    }
    // We can finally actually set up the plan
    printf("axis 1, get plan r2c\n");
    if (hipfftMakePlan1d(*plan, ncols, HIPFFT_R2C, nrows, work_size)!=HIPFFT_SUCCESS)
        fprintf(stderr,"Error planning dft.\n");
    printf("the plan in C has value %d\n", *plan);
    printf("plan worksize in C has value %d\n", *work_size);
    printf("plan worksize in C has size %d\n", sizeof(work_size));
}
/*--------------------------------------------------------------------------------*/
void get_plan_c2r(int nrows, int ncols, hipfftHandle *plan, size_t * work_size)
{
    // nrows, ncols of output
    // Initialize an empty plan.
    hipError_t hipError_t;

        hipError_t = hipGetLastError();

        if( hipError_t != hipSuccess )

        {

        fprintf(stderr, "CUDA Runtime API Error reported : %s\n", hipGetErrorString(hipError_t));

        exit(EXIT_FAILURE);

        }
    hipfftCreate(plan);
    // Turn off auto-allocation by default. This must be done before the
    // plan is actually created, so we can't use the shortcut hipfftPlan1d
    hipfftResult_t error;
    error=hipfftSetAutoAllocation(*plan, 0);
    if(error!=HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT setAutoAllocation returned %d", error);
    }
    printf("axis 1, get plan c2r\n");
    // We can finally actually set up the plan
    if (hipfftMakePlan1d(*plan, ncols, HIPFFT_C2R, nrows, work_size)!=HIPFFT_SUCCESS)
        fprintf(stderr,"Error planning dft.\n");
    printf("the plan in C has value %d\n", *plan);
    printf("plan worksize in C has value %d\n", *work_size);
    printf("plan worksize in C has size %d\n", sizeof(work_size));
}
void get_plan_c2c(int nrows, int ncols, hipfftHandle *plan, size_t * work_size)
{
    // nrows -> batch, ncols -> size
    // Initialize an empty plan.
    hipError_t hipError_t;

        hipError_t = hipGetLastError();

        if( hipError_t != hipSuccess )

        {

        fprintf(stderr, "CUDA Runtime API Error reported : %s\n", hipGetErrorString(hipError_t));

        exit(EXIT_FAILURE);

        }
    hipfftCreate(plan);
    // Turn off auto-allocation by default. This must be done before the
    // plan is actually created, so we can't use the shortcut hipfftPlan1d
    hipfftResult_t error;
    error=hipfftSetAutoAllocation(*plan, 0);
    if(error!=HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT setAutoAllocation returned %d", error);
    }
    printf("axis 1, get plan c2c\n");
    // We can finally actually set up the plan
    if (hipfftMakePlan1d(*plan, ncols, HIPFFT_C2C, nrows, work_size)!=HIPFFT_SUCCESS)
        fprintf(stderr,"Error planning dft.\n");
    printf("the plan in C has value %d\n", *plan);
    printf("plan worksize in C has value %d\n", *work_size);
    printf("plan worksize in C has size %d\n", sizeof(work_size));
}
        
/*--------------------------------------------------------------------------------*/
void destroy_plan(hipfftHandle *plan)
{
  if (hipfftDestroy(*plan)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan.\n");
}
void get_plan_size(hipfftHandle *plan, size_t *sz)
{
  if (hipfftGetSize(*plan,sz)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error querying plan size.\n");
}
/*--------------------------------------------------------------------------------*/
void set_plan_scratch(hipfftHandle *plan, void *buf)
{
  if (hipfftSetWorkArea(*plan,buf)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error assigning buffer in set_plan_scratch.\n");
  //else
  //printf("successfully assigned buffer.\n");
          
}

/*================================================================================*/
}