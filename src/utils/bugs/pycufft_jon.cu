//nvcc -o libpycufft.so pycufft.cu -shared -lcufft -Xcompiler -fPIC -lgomp

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <omp.h>

extern "C" void get_work_sizes_r2c(long int *sz, int nsize, long int *nbytes)
{
  size_t nb_max=0;
  for (int i=0;i<nsize;i++)
    {
      hipfftHandle plan;
      if (i==0)
        printf("plan size is %ld\n",sizeof(hipfftHandle));
      int n=sz[2*i];
      int ntrans=sz[2*i+1];
      int rank=1; //we're doing 1D transforms
      int nn=(n/2)+1;
      int istride=1;
      int idist=nn;
      int oembed=nn;
      if (hipfftPlanMany(&plan, rank, &n, &nn, istride, idist,&oembed, istride,oembed, HIPFFT_R2C,ntrans)!=HIPFFT_SUCCESS) {
        fprintf(stderr,"Error in planning r2c with dimensions %d %d\n",n,ntrans);
        *nbytes=-1;
        return;
          
      }
      hipfftSetAutoAllocation(plan,1);
      size_t nb;
      if (hipfftGetSize(plan,&nb)!=HIPFFT_SUCCESS) {
        fprintf(stderr,"Error in querying size wth dimensions %d %d\n",n,ntrans);
        *nbytes=-1;
        return;
      }
      if (nb>nb_max)
        nb_max=nb;
      if (hipfftDestroy(plan)!= HIPFFT_SUCCESS) {
        fprintf(stderr,"Error destroying plan.\n");
        *nbytes=-1;
        return;
      }
    }
}
/*--------------------------------------------------------------------------------*/

void cufft_c2r(float *out, hipfftComplex *data, int len, int ntrans)
{
  //float *out;
  hipfftHandle plan;
  
  if (hipfftPlan1d(&plan,len,HIPFFT_C2R, ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning dft\n");
  //hipDeviceSynchronize();
  //double t1=omp_get_wtime();
  if (hipfftExecC2R(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing dft\n");
  //hipDeviceSynchronize();
  //double t2=omp_get_wtime();
  //printf("took %12.4g seconds to do fft.\n",t2-t1);

  if (hipfftDestroy(plan)!= HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan.\n");
}
/*--------------------------------------------------------------------------------*/
void cufft_c2r_wplan(float *out, hipfftComplex *data, hipfftHandle plan)
{
  if (hipfftExecC2R(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing idft\n");
  hipDeviceSynchronize();
}
/*--------------------------------------------------------------------------------*/
void cufft_c2r_columns(float *out, hipfftComplex *data,int len, int ntrans)
{
  hipfftHandle plan;
  int rank=1;
  int inembed[rank] = {ntrans};
  int onembed[rank]={ntrans};
  int istride=ntrans;
  int idist=1;
  int ostride=ntrans;
  int odist=1;
  if (hipfftPlanMany(&plan,rank,&len,inembed,istride,idist,onembed,ostride,odist,HIPFFT_C2R,ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning DFT in c2r_columns.\n");
  if (hipfftExecC2R(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing DFT in c2r_columns.\n");
  if (hipfftDestroy(plan)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan in c2r_columns.\n");

}

/*--------------------------------------------------------------------------------*/
extern "C" void cufft_c2r_host(float *out, hipfftComplex *data, int n, int m, int axis)
{
  float *dout;
  hipfftComplex *din;
  if (hipMalloc((void **)&din,sizeof(hipfftComplex)*n*m)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");
  if (hipMemcpy(din,data,n*m*sizeof(hipfftComplex),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying data to device.\n");
  if (axis==0) {
    if (hipMalloc((void **)&dout,sizeof(float)*n*m)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_c2r_columns(dout,din,n,m);
    if (hipMemcpy(out,dout,sizeof(float)*n*m,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in c2r\n");
  }
  else {
    if (hipMalloc((void **)&dout,sizeof(float)*n*m)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_c2r(dout,din,m,n);
    if (hipMemcpy(out,dout,sizeof(float)*m*n,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in c2r\n");

  }
}

/*--------------------------------------------------------------------------------*/
void cufft_r2c(hipfftComplex *out, float *data, int len, int ntrans)
{
  hipfftHandle plan;
  hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    hipEventRecord(start, 0);
  if (hipfftPlan1d(&plan,len,HIPFFT_R2C, ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning dft\n");
  
//   double t1=omp_get_wtime();
//     // 
  if (hipfftExecR2C(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing dft\n");
    
  if (hipfftDestroy(plan)!= HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan.\n");
    // hipDeviceSynchronize();
//   double t2=omp_get_wtime();
//   printf("r2c took %12.4g\n",t2-t1);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("r2c took %12.4g\n",milliseconds);
}
/*--------------------------------------------------------------------------------*/
void cufft_r2c_wplan(hipfftComplex *out, float *data, int len, int ntrans,hipfftHandle plan)
{
  if (hipfftExecR2C(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing dft\n");
}

/*--------------------------------------------------------------------------------*/
void cufft_r2c_columns(hipfftComplex *out, float *data, int len, int ntrans)
{
  hipfftHandle plan;
  int rank=1;
  int inembed[rank] = {len};
  int onembed[rank]={ntrans};
  int istride=ntrans;
  int idist=1;
  int ostride=ntrans;
  int odist=1;
  if (hipfftPlanMany(&plan,rank,&len,inembed,istride,idist,onembed,ostride,odist,HIPFFT_R2C,ntrans)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error planning DFT in r2c_columns.\n");
  if (hipfftExecR2C(plan,data,out)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error executing DFT in r2c_columns.\n");
  if (hipfftDestroy(plan)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan in r2c_columns.\n");
  
}


/*--------------------------------------------------------------------------------*/
extern "C" {

void cufft_r2c_gpu(hipfftComplex *out, float *data, int n, int m, int axis)
{
  if (axis==1)
    cufft_r2c(out,data,m,n);
  else
    cufft_r2c_columns(out,data,n,m);
}
/*--------------------------------------------------------------------------------*/

void cufft_r2c_gpu_wplan(hipfftComplex *out, float *data, int n, int m, int axis,hipfftHandle *plan)
{
  if (axis==1)
    cufft_r2c_wplan(out,data,m,n,*plan);
  else
    cufft_r2c_columns(out,data,n,m);
}
/*--------------------------------------------------------------------------------*/

void cufft_c2r_gpu_wplan(float  *out, hipfftComplex *data, hipfftHandle *plan)
{
  cufft_c2r_wplan(out,data,*plan);
}
/*--------------------------------------------------------------------------------*/

void cufft_c2r_gpu(float *out, hipfftComplex *data, int n, int m, int axis)
{
  if (axis==1)
    cufft_c2r(out,data,m,n);
  else
    cufft_c2r_columns(out,data,n,m);
}
/*--------------------------------------------------------------------------------*/
void get_plan_size(hipfftHandle *plan, size_t *sz)
{
  if (hipfftGetSize(*plan,sz)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error querying plan size.\n");
}
/*--------------------------------------------------------------------------------*/
void get_plan_r2c(int n, int m, int axis, hipfftHandle *plan, int alloc)
{
  if (axis==1) {
    // Initialize an empty plan.
    hipfftCreate(plan);
    // Turn off auto-allocation if required. This must be done before the
    // plan is actually created, so we can't use the shortcut hipfftPlan1d
    if (!alloc) {
        hipfftSetAutoAllocation(*plan, 0);
        //hipfftSetWorkArea(*plan, NULL);
    }
    // We can finally actually set up the plan
    size_t work_size;
    if (hipfftMakePlan1d(*plan, m, HIPFFT_R2C, n, &work_size)!=HIPFFT_SUCCESS)
      fprintf(stderr,"Error planning dft.\n");
  } else {
    // SKN: I assume this is TODO?
  }
}
        
/*--------------------------------------------------------------------------------*/
void get_plan_c2r(int n, int m, int axis,hipfftHandle *plan, int alloc)
//make sure n and m correspond to the size of the *output* transform
{
  if (axis==1) {
    // Initialize an empty plan.
    hipfftCreate(plan);
    // Turn off auto-allocation if required. This must be done before the
    // plan is actually created, so we can't use the shortcut hipfftPlan1d
    if (!alloc) {
        hipfftSetAutoAllocation(*plan, 0);
        //hipfftSetWorkArea(*plan, NULL);
    }
    // We can finally actually set up the plan
    size_t work_size;
    if (hipfftMakePlan1d(*plan, m, HIPFFT_C2R, n, &work_size)!=HIPFFT_SUCCESS)
      fprintf(stderr,"Error planning dft.\n");
  } else {
    // SKN: I assume this is TODO?
  }
}
        
/*--------------------------------------------------------------------------------*/
void destroy_plan(hipfftHandle *plan)
{
  if (hipfftDestroy(*plan)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error destroying plan.\n");
}
        
/*--------------------------------------------------------------------------------*/
void set_plan_scratch(hipfftHandle plan,void *buf)
{
  if (hipfftSetWorkArea(plan,buf)!=HIPFFT_SUCCESS)
    fprintf(stderr,"Error assigning buffer in set_plan_scratch.\n");
  //else
  //printf("successfully assigned buffer.\n");
          
}

/*--------------------------------------------------------------------------------*/
void cufft_r2c_host(hipfftComplex *out, float *data, int n, int m, int axis)
{
  hipfftComplex *dout;
  float *din;
  int nn;
  if (axis==0)
    nn=n/2+1;
  else
    nn=m/2+1;
  if (hipMalloc((void **)&din,sizeof(float)*n*m)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");
  if (hipMemcpy(din,data,n*m*sizeof(float),hipMemcpyHostToDevice)!=hipSuccess)
    fprintf(stderr,"Error copying data to device.\n");
  if (axis==0) {
    if (hipMalloc((void **)&dout,sizeof(hipfftComplex)*nn*m)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_r2c_columns(dout,din,n,m);
    //printf("copying %d %d\n",nn,m);
    if (hipMemcpy(out,dout,sizeof(hipfftComplex)*nn*m,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in r2c\n");
  }
  else {
    if (hipMalloc((void **)&dout,sizeof(hipfftComplex)*n*nn)!=hipSuccess)
      fprintf(stderr,"error in hipMalloc\n");
    cufft_r2c(dout,din,m,n);
    //printf("copying %d %d\n",n,nn);
    if (hipMemcpy(out,dout,sizeof(hipfftComplex)*nn*n,hipMemcpyDeviceToHost)!=hipSuccess)
      fprintf(stderr,"Error copying result to host in r2c\n");
  
  }
}

}



/*================================================================================*/


#if 0

int main(int argc, char *argv[])
{
  printf("Hello world!\n");
  int ndet=1000;
  int nsamp=1<<18;
  printf("nsamp is %d\n",nsamp);

  float *fdat=(float *)malloc(sizeof(float)*ndet*nsamp);
  if (fdat!=NULL)
    printf("successfully malloced array on host.\n");

  float *ddat;
  if (hipMalloc((void **)&ddat,sizeof(float)*nsamp*ndet)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");
  hipComplex *dtrans;
  if (hipMalloc((void **)&dtrans,sizeof(hipComplex)*nsamp*ndet)!=hipSuccess)
    fprintf(stderr,"error in hipMalloc\n");

  
  
}
#endif